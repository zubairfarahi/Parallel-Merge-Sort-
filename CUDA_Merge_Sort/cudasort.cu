
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>

#define THREADS 512
#ifdef __cplusplus
__global__ void mergesort(float* source, float* dest, long size, long width, long slices, dim3* threads, dim3* blocks)
{

	unsigned int x=threads->x;
	unsigned int y=threads->y;
	unsigned int z=threads->z;
	unsigned int idx = threadIdx.x + (x*threadIdx.y)+(x*y*threadIdx.z) +(x*y*z*blockIdx.x) +(x*y*z*z*blockIdx.y)+(x*y*z*y*blockIdx.z);
	long count=0,start = width*idx*slices,middle,end;

	while(count < slices)
	{
        	if (start >= size)
	            break;

        	middle = min(start + (width >> 1), size);
	        end = min(start + width, size);
     
		long i = start, j = middle,k=start;
		while(k < end) 
		{
		        if (i < middle && (j >= end || source[i] < source[j]))
			{
		            dest[k] = source[i];
		            i++;
		        }
			else
			{
		            dest[k] = source[j];
		            j++;
		        }
			k++;
		}

	        start =start+width;
		count++;
	}
}

extern "C"
{
#endif

int cuda_sort(int number_of_elements, float *a)
{
	dim3 thread_units,block_units;
/*	printf("\n Before Sort \n");
        for(int i=0;i<number_of_elements;i++)
        {
                printf("%f \n",a[i]);
        }
*/
//	int ret=(number_of_elements / 2)%512;
	thread_units.x = THREADS;
	thread_units.y = 1;
	thread_units.z = 1;

	block_units.x = THREADS/2;//(ret==0)?number_of_elements/2:number_of_elements/2+512-ret;
	block_units.y = 1;
	block_units.z = 1;

	float *in,*out;
        dim3 *threads,*blocks;
  
//	printf("INSIDE CUDA SORT\n");
	hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 16);
	hipMalloc((void**)&in,number_of_elements*sizeof(float));
	hipMalloc((void**)&out,number_of_elements*sizeof(float));
	hipMalloc((void**) &threads, sizeof(dim3));
	hipMalloc((void**) &blocks, sizeof(dim3));
	
	hipMemcpy(in,a, number_of_elements*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(threads, &thread_units, sizeof(dim3), hipMemcpyHostToDevice);
	hipMemcpy(blocks, &block_units, sizeof(dim3), hipMemcpyHostToDevice);
	long nThreads = THREADS*(THREADS/2);
	float *data1 = in,*data2 = out;

	for (int i = 2; i < (number_of_elements << 1); i <<= 1) 
	{
        	long slices = number_of_elements / ((nThreads) * i) + 1;
		mergesort<<<block_units,thread_units>>>(data1, data2, number_of_elements, i, slices, threads, blocks);
	//swapping	
		if(data1==in)			
			data1 = out;
		else
			data1=in;
		if(data2==in)
		        data2 = out;
		else 
			data2=in;
	}
	hipMemcpy(a, data1, number_of_elements * sizeof(float), hipMemcpyDeviceToHost);
/*	printf("\n After Sort \n");
	for(int i=0;i<number_of_elements;i++)
	{
		printf("%f \n",a[i]);
	}*/
	hipFree(out);
	hipFree(in);
	hipDeviceReset();
	return 0;
}

#ifdef __cplusplus
}
#endif
